#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <string.h>
#include <thrust/generate.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <algorithm>
#include <cstdlib>
#include <stdio.h>
#include <stdlib.h>
#include <Windows.h>


// Must be run on Windows

int main(void)
{
    LARGE_INTEGER StartingTime, EndingTime, ElapsedMicroseconds;
    LARGE_INTEGER Frequency;

    // Set N For Each Test here
    int N = 50000000;
    bool runHost = true;

    std::string hostResult;
    if (runHost) {
        // Run on the Host Serially
        thrust::host_vector<int> h_vec(N);
        thrust::generate(h_vec.begin(), h_vec.end(), rand);

        QueryPerformanceFrequency(&Frequency);
        QueryPerformanceCounter(&StartingTime);

        thrust::sort(h_vec.begin(), h_vec.end());

        QueryPerformanceCounter(&EndingTime);
        ElapsedMicroseconds.QuadPart = EndingTime.QuadPart - StartingTime.QuadPart;
        ElapsedMicroseconds.QuadPart *= 1000000;
        ElapsedMicroseconds.QuadPart /= Frequency.QuadPart;

        hostResult = "The elapsed time was " + std::to_string(ElapsedMicroseconds.QuadPart) + " Microseconds.\n";
    }
    else {
        hostResult = "Host version not run.\n";
    }
    // GPU VERSION

    thrust::host_vector<int> h_vec2(N);

    LARGE_INTEGER StartingTime2, EndingTime2, ElapsedMicroseconds2;
    LARGE_INTEGER Frequency2;
    QueryPerformanceFrequency(&Frequency2);
    QueryPerformanceCounter(&StartingTime2);

    // transfer data to the device
    thrust::device_vector<int> d_vec = h_vec2;
    thrust::generate(h_vec2.begin(), h_vec2.end(), rand);

    // use thrust to sort the device array.
    thrust::sort(d_vec.begin(), d_vec.end());
  
    // transfer data back to host
    thrust::copy(d_vec.begin(), d_vec.end(), h_vec2.begin());

    QueryPerformanceCounter(&EndingTime2);
    ElapsedMicroseconds2.QuadPart = EndingTime2.QuadPart - StartingTime2.QuadPart;
    ElapsedMicroseconds2.QuadPart *= 1000000;
    ElapsedMicroseconds2.QuadPart /= Frequency2.QuadPart;

    std::string deviceResult = "The elapsed time for GPU Sort was " + std::to_string(ElapsedMicroseconds2.QuadPart) + " Microseconds.\n";

    std::cout << hostResult << deviceResult;

    return 0;
}
