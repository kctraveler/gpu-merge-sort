#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <string.h>
#include <thrust/generate.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <algorithm>
#include <cstdlib>
#include <stdio.h>
#include <stdlib.h>
#include <Windows.h>

/*
Unlike the other c++ example, this must be run on windows. I had issues with the NVCC compiler on the LUC cluster and decided to use my local workstation. In order to get a timer for profiling I had to use a Windows library.
*/

int main(void)
{
    LARGE_INTEGER StartingTime, EndingTime, ElapsedMicroseconds;
    LARGE_INTEGER Frequency;

    // Create host vector prior to starting timer
    thrust::host_vector<double> h_vec(16777216);
    thrust::generate(h_vec.begin(), h_vec.end(), rand);

    // Start timer for profiling
    QueryPerformanceFrequency(&Frequency);
    QueryPerformanceCounter(&StartingTime);

    // transfer data to the device
    thrust::device_vector<double> d_vec = h_vec;
    
    // use thrust to sort the device array.
    thrust::sort(d_vec.begin(), d_vec.end());
    // sort on the host
    //std::sort(h_vec.begin(), h_vec.end());

    // transfer data back to host
    thrust::copy(d_vec.begin(), d_vec.end(), h_vec.begin());

    // End timer and calculat Microseconds
    QueryPerformanceCounter(&EndingTime);
    ElapsedMicroseconds.QuadPart = EndingTime.QuadPart - StartingTime.QuadPart;
    ElapsedMicroseconds.QuadPart *= 1000000;
    ElapsedMicroseconds.QuadPart /= Frequency.QuadPart;
    std::cout << "The elapsed time was " <<  ElapsedMicroseconds.QuadPart;

    return 0;
}